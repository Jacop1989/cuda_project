#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100000  // จำนวนจุดใน 3D space

// CUDA kernel สำหรับ 3D transformations (Rotation และ Translation)
__global__ void transform_points(float *x, float *y, float *z, int n, float theta, float tx, float ty, float tz) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        // การหมุนรอบแกน z-axis (2D rotation)
        float new_x = x[idx] * cos(theta) - y[idx] * sin(theta);
        float new_y = x[idx] * sin(theta) + y[idx] * cos(theta);

        // การย้ายตำแหน่ง (Translation)
        x[idx] = new_x + tx;
        y[idx] = new_y + ty;
        z[idx] = z[idx] + tz;
    }
}

int main() {
    size_t size = N * sizeof(float);

    // จองหน่วยความจำบน Host สำหรับพิกัด (x, y, z)
    float *h_x = (float *)malloc(size);
    float *h_y = (float *)malloc(size);
    float *h_z = (float *)malloc(size);

    // กำหนดค่าเริ่มต้นในพิกัด
    for (int i = 0; i < N; i++) {
        h_x[i] = i * 0.1f;
        h_y[i] = i * 0.1f;
        h_z[i] = i * 0.1f;
    }

    // จองหน่วยความจำบน Device สำหรับพิกัด (x, y, z)
    float *d_x, *d_y, *d_z;
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);

    // คัดลอกข้อมูลจาก Host ไปยัง Device
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, size, hipMemcpyHostToDevice);

    // กำหนดพารามิเตอร์การหมุนและการย้ายตำแหน่ง
    float theta = 3.14159 / 4;  // หมุน 45 องศา
    float tx = 1.0f, ty = 2.0f, tz = 3.0f;  // การย้ายตำแหน่ง

    // กำหนดจำนวนบล็อกและเทรด
    int threads_per_block = 256;
    int number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    // จับเวลาบน GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    transform_points<<<number_of_blocks, threads_per_block>>>(d_x, d_y, d_z, N, theta, tx, ty, tz);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Time for 3D Transformations: %f ms\n", milliseconds);

    // คัดลอกผลลัพธ์กลับมายัง Host
    hipMemcpy(h_x, d_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);

    // แสดงพิกัดบางส่วน
    for (int i = 0; i < 5; i++) {
        printf("Point %d -> (x: %f, y: %f, z: %f)\n", i, h_x[i], h_y[i], h_z[i]);
    }

    // ลบหน่วยความจำ
    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}
