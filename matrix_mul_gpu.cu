#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000  // ขนาดของเมทริกซ์

// CUDA kernel สำหรับการคูณเมทริกซ์
__global__ void matrix_mul(float *a, float *b, float *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0;
        for (int i = 0; i < n; i++) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    size_t size = N * N * sizeof(float);

    // จองหน่วยความจำบน Host
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // กำหนดค่าเริ่มต้นในเมทริกซ์
    for (int i = 0; i < N * N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // จองหน่วยความจำบน Device
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // คัดลอกข้อมูลจาก Host ไปยัง Device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // กำหนดจำนวนบล็อกและเทรด (จาก 16x16 เป็น 32x32)
    dim3 threads_per_block(32, 32);
    dim3 number_of_blocks((N + threads_per_block.x - 1) / threads_per_block.x, 
                          (N + threads_per_block.y - 1) / threads_per_block.y);

    // จับเวลาบน GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrix_mul<<<number_of_blocks, threads_per_block>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Time for Matrix Multiplication with 32x32 blocks: %f ms\n", milliseconds);

    // คัดลอกผลลัพธ์กลับมายัง Host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // ลบหน่วยความจำ
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
