#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define WIDTH  800  // ความกว้างของภาพ
#define HEIGHT 600  // ความสูงของภาพ

// โครงสร้างสำหรับสี
struct Color {
    float r, g, b;
};

// โครงสร้างสำหรับเวกเตอร์ 3 มิติ
struct Vec3 {
    float x, y, z;

    __device__ Vec3 operator+(const Vec3& v) const {
        return Vec3{x + v.x, y + v.y, z + v.z};
    }

    __device__ Vec3 operator-(const Vec3& v) const {
        return Vec3{x - v.x, y - v.y, z - v.z};
    }

    __device__ Vec3 operator*(float t) const {
        return Vec3{x * t, y * t, z * t};
    }

    __device__ float dot(const Vec3& v) const {
        return x * v.x + y * v.y + z * v.z;
    }

    __device__ Vec3 normalize() const {
        float length = sqrtf(x * x + y * y + z * z);
        return Vec3{x / length, y / length, z / length};
    }
};

// Function สำหรับตรวจสอบการชนระหว่าง Ray กับทรงกลม
__device__ bool hit_sphere(const Vec3& sphere_center, float sphere_radius, const Vec3& ray_origin, const Vec3& ray_direction) {
    Vec3 oc = ray_origin - sphere_center;
    float a = ray_direction.dot(ray_direction);
    float b = 2.0f * oc.dot(ray_direction);
    float c = oc.dot(oc) - sphere_radius * sphere_radius;
    float discriminant = b * b - 4 * a * c;
    return (discriminant > 0);
}

// Function สำหรับ trace ray ใน scene ที่มีหลายทรงกลม
__device__ Color trace_ray(const Vec3& ray_origin, const Vec3& ray_direction) {
    Vec3 sphere_center1 = {0.0f, 0.0f, -1.0f};  // ทรงกลมแรก
    Vec3 sphere_center2 = {0.5f, 0.0f, -1.5f};  // ทรงกลมที่สอง
    float sphere_radius1 = 0.5f;
    float sphere_radius2 = 0.3f;

    // ตรวจสอบการชนระหว่าง Ray กับทรงกลมแรก
    if (hit_sphere(sphere_center1, sphere_radius1, ray_origin, ray_direction)) {
        return Color{1.0f, 0.0f, 0.0f};  // สีแดงสำหรับทรงกลมแรก
    }
    
    // ตรวจสอบการชนระหว่าง Ray กับทรงกลมที่สอง
    if (hit_sphere(sphere_center2, sphere_radius2, ray_origin, ray_direction)) {
        return Color{0.0f, 1.0f, 0.0f};  // สีเขียวสำหรับทรงกลมที่สอง
    }

    // พื้นหลังถ้าไม่มีการชน
    return Color{0.5f, 0.7f, 1.0f};  // พื้นหลังสีฟ้า
}

// CUDA kernel สำหรับ Ray Tracing พร้อมวัตถุหลายชิ้น
__global__ void ray_tracing(Color *image) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < WIDTH && y < HEIGHT) {
        int idx = y * WIDTH + x;

        // คำนวณพิกัดของรังสี (Ray) ออกจากกล้อง
        float u = float(x) / WIDTH;
        float v = float(y) / HEIGHT;
        Vec3 ray_origin = {0.0f, 0.0f, 0.0f};  // กล้องอยู่ที่ (0, 0, 0)
        Vec3 ray_direction = {2.0f * u - 1.0f, 2.0f * v - 1.0f, -1.0f};  // รังสีออกจากกล้อง
        ray_direction = ray_direction.normalize();

        // คำนวณสีของพิกเซล
        image[idx] = trace_ray(ray_origin, ray_direction);
    }
}

int main() {
    size_t size = WIDTH * HEIGHT * sizeof(Color);

    // จองหน่วยความจำบน Host สำหรับภาพ
    Color *h_image = (Color *)malloc(size);

    // จองหน่วยความจำบน Device สำหรับภาพ
    Color *d_image;
    hipMalloc(&d_image, size);

    // กำหนดจำนวนบล็อกและเทรด
    dim3 threads_per_block(16, 16);
    dim3 number_of_blocks((WIDTH + threads_per_block.x - 1) / threads_per_block.x, 
                          (HEIGHT + threads_per_block.y - 1) / threads_per_block.y);

    // จับเวลาบน GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    ray_tracing<<<number_of_blocks, threads_per_block>>>(d_image);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Time for Ray Tracing with Multiple Objects: %f ms\n", milliseconds);

    // คัดลอกผลลัพธ์กลับมายัง Host
    hipMemcpy(h_image, d_image, size, hipMemcpyDeviceToHost);

    // เขียนผลลัพธ์ภาพลงไฟล์ (PPM format)
    FILE *f = fopen("ray_tracing_multi_object.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", WIDTH, HEIGHT);
    for (int i = 0; i < WIDTH * HEIGHT; i++) {
        int r = int(255.99 * h_image[i].r);
        int g = int(255.99 * h_image[i].g);
        int b = int(255.99 * h_image[i].b);
        fprintf(f, "%d %d %d\n", r, g, b);
    }
    fclose(f);

    // ลบหน่วยความจำ
    free(h_image);
    hipFree(d_image);

    return 0;
}
