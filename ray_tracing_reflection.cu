#include <stdio.h>
#include <hip/hip_runtime.h>

#define WIDTH  800  // ความกว้างของภาพ
#define HEIGHT 600  // ความสูงของภาพ
#define MAX_DEPTH 3  // ความลึกสูงสุดของการสะท้อน

// โครงสร้างสำหรับสี
struct Color {
    float r, g, b;
};

// โครงสร้างสำหรับเวกเตอร์ 3 มิติ
struct Vec3 {
    float x, y, z;

    __device__ Vec3 operator+(const Vec3& v) const {
        return Vec3{x + v.x, y + v.y, z + v.z};
    }

    __device__ Vec3 operator-(const Vec3& v) const {
        return Vec3{x - v.x, y - v.y, z - v.z};
    }

    __device__ Vec3 operator*(float t) const {
        return Vec3{x * t, y * t, z * t};
    }

    __device__ float dot(const Vec3& v) const {
        return x * v.x + y * v.y + z * v.z;
    }

    __device__ Vec3 normalize() const {
        float length = sqrtf(x * x + y * y + z * z);
        return Vec3{x / length, y / length, z / length};
    }
};

// CUDA kernel สำหรับการทำ Ray Tracing พร้อม Reflection
__device__ Color trace_ray(const Vec3& ray_origin, const Vec3& ray_direction, Vec3 sphere_center, float sphere_radius, int depth) {
    Color black = {0.0f, 0.0f, 0.0f};
    if (depth > MAX_DEPTH) return black;

    // คำนวณการชนกับทรงกลม (Ray-Sphere Intersection)
    Vec3 oc = ray_origin - sphere_center;
    float a = ray_direction.dot(ray_direction);
    float b = 2.0f * oc.dot(ray_direction);
    float c = oc.dot(oc) - sphere_radius * sphere_radius;
    float discriminant = b * b - 4 * a * c;

    if (discriminant > 0) {
        // รังสีชนกับทรงกลม
        float t = (-b - sqrtf(discriminant)) / (2.0f * a);
        Vec3 hit_point = ray_origin + ray_direction * t;
        Vec3 normal = (hit_point - sphere_center).normalize();

        // คำนวณทิศทางของรังสีที่สะท้อน
        Vec3 reflection_dir = ray_direction - normal * 2.0f * ray_direction.dot(normal);
        reflection_dir = reflection_dir.normalize();

        // สีของการสะท้อน
        Color reflection_color = trace_ray(hit_point, reflection_dir, sphere_center, sphere_radius, depth + 1);

        // การรวมสีของการชนกับสีของการสะท้อน
        Color hit_color = {1.0f, 0.0f, 0.0f};  // สีของทรงกลม
        hit_color.r = hit_color.r * 0.8f + reflection_color.r * 0.2f;
        hit_color.g = hit_color.g * 0.8f + reflection_color.g * 0.2f;
        hit_color.b = hit_color.b * 0.8f + reflection_color.b * 0.2f;

        return hit_color;
    }

    // พื้นหลัง
    Color background_color = {0.5f, 0.7f, 1.0f};  // สีพื้นหลัง (ฟ้า)
    return background_color;
}

__global__ void ray_tracing(Color *image, Vec3 sphere_center, float sphere_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < WIDTH && y < HEIGHT) {
        int idx = y * WIDTH + x;

        // คำนวณพิกัดของรังสี (Ray) ออกจากกล้อง
        float u = float(x) / WIDTH;
        float v = float(y) / HEIGHT;
        Vec3 ray_origin = {0.0f, 0.0f, 0.0f};  // กล้องอยู่ที่ (0, 0, 0)
        Vec3 ray_direction = {2.0f * u - 1.0f, 2.0f * v - 1.0f, -1.0f};  // รังสีออกจากกล้อง
        ray_direction = ray_direction.normalize();

        // คำนวณสีของพิกเซล
        image[idx] = trace_ray(ray_origin, ray_direction, sphere_center, sphere_radius, 0);
    }
}

int main() {
    size_t size = WIDTH * HEIGHT * sizeof(Color);

    // จองหน่วยความจำบน Host สำหรับภาพ
    Color *h_image = (Color *)malloc(size);

    // จองหน่วยความจำบน Device สำหรับภาพ
    Color *d_image;
    hipMalloc(&d_image, size);

    // กำหนดพารามิเตอร์ของทรงกลม
    Vec3 sphere_center = {0.0f, 0.0f, -1.0f};  // ทรงกลมอยู่ที่ (0, 0, -1)
    float sphere_radius = 0.5f;  // รัศมีของทรงกลม

    // กำหนดจำนวนบล็อกและเทรด
    dim3 threads_per_block(16, 16);
    dim3 number_of_blocks((WIDTH + threads_per_block.x - 1) / threads_per_block.x, 
                          (HEIGHT + threads_per_block.y - 1) / threads_per_block.y);

    // จับเวลาบน GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    ray_tracing<<<number_of_blocks, threads_per_block>>>(d_image, sphere_center, sphere_radius);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Time for Ray Tracing with Reflection: %f ms\n", milliseconds);

    // คัดลอกผลลัพธ์กลับมายัง Host
    hipMemcpy(h_image, d_image, size, hipMemcpyDeviceToHost);

    // เขียนผลลัพธ์ภาพลงไฟล์ (PPM format)
    FILE *f = fopen("ray_tracing_reflection.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", WIDTH, HEIGHT);
    for (int i = 0; i < WIDTH * HEIGHT; i++) {
        int r = int(255.99 * h_image[i].r);
        int g = int(255.99 * h_image[i].g);
        int b = int(255.99 * h_image[i].b);
        fprintf(f, "%d %d %d\n", r, g, b);
    }
    fclose(f);

    // ลบหน่วยความจำ
    free(h_image);
    hipFree(d_image);

    return 0;
}
