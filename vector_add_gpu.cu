#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100000000  // เพิ่มขนาดเวกเตอร์

// CUDA kernel สำหรับการบวกเวกเตอร์
__global__ void vector_add(float *a, float *b, float *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    float *h_a = (float *)malloc(N * sizeof(float));
    float *h_b = (float *)malloc(N * sizeof(float));
    float *h_c = (float *)malloc(N * sizeof(float));

    // กำหนดค่าเริ่มต้น
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // จองหน่วยความจำบน Device
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // คัดลอกข้อมูลจาก Host ไปยัง Device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // จับเวลาบน GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threads_per_block = 256;
    int number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    hipEventRecord(start);
    vector_add<<<number_of_blocks, threads_per_block>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Time: %f ms\n", milliseconds);

    // คัดลอกผลลัพธ์กลับมายัง Host
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // ลบหน่วยความจำ
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
